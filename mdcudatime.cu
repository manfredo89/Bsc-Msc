/* Programma di simulazione di dinamica molecolare */
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define Rcut 2.5f // cutoff distance
#define DIM 3
#define dt 0.0005f
#define eCut -0.01631689114f // 4 * ((1 / ( pow (Rcut, 12))) - (1 / pow(Rcut,6))) // cutoff energy
#define mass 1.0f
#define PI 3.1415926535f
#define NUM_THREAD 64  // Number of threads per block
#define NUM_BLOCK (int) ceil (Natoms/(float)NUM_THREAD)  // Numb of thread blocks

typedef struct {
    
    float x, y, z; // coordinates
    float w; // free parameter
    
    } M_double4;


/****************************************************************************************/
/* first half kick for Verlet integration */

__global__ void HalfStep (M_double4 *v, M_double4 *a, int Natoms){
    
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    
    if (i < Natoms){
        
        v[i].x += 0.5f * dt * a[i].x;
        v[i].y += 0.5f * dt * a[i].y;
        v[i].z += 0.5f * dt * a[i].z;
        v[i].w = 0.5f * mass * ((v[i].x * v[i].x) + (v[i].y * v[i].y) + (v[i].z * v[i].z)); // kinetic energy
    }
}
/****************************************************************************************/
/* positions updates */

__global__ void Position (M_double4 *r, M_double4 *v, M_double4 *a, int Natoms){
    
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    
    if (i < Natoms){
        
        v[i].x += 0.5f * dt * a[i].x;
        v[i].y += 0.5f * dt * a[i].y;
        v[i].z += 0.5f * dt * a[i].z;
        
        r[i].x += v[i].x * dt;
        r[i].y += v[i].y * dt;
        r[i].z += v[i].z * dt;
        
        }
    
}
/****************************************************************************************/
/* the function generates normal distributed number with Box-Muller algorithm */

float Gaussian (void){
    
    float x, y, s;
    
    x = ((float)lrand48()/RAND_MAX);
    y = ((float)lrand48()/RAND_MAX);
    s = sqrt (-2.0f * log(x)) * cos (2.0f * PI * y);
    
    return s;
    
}

/****************************************************************************************/
/* calculus of forces with Lennard Jones potential and cutoff energy (see Molecular Simulation by D. Frenkel) */

__global__ void Acceleration(M_double4 *r, M_double4 *a, int Natoms, float rho){
    
    
    int i, j, axis;
    float d2, d2inv, d6inv, f, dr[DIM];
    float L = powf(Natoms / rho, 1.0f/3);
    
    i = blockIdx.x*blockDim.x+threadIdx.x;
     
     if (i < Natoms){
    
        a[i].x = 0.0f;
        a[i].y = 0.0f;
        a[i].z = 0.0f;
        a[i].w = 0.0f;
       
        for (j=0;j<Natoms;j++){
            if (i == j) continue;
            
            	    dr[0] = r[i].x - r[j].x;
		    dr[1] = r[i].y - r[j].y;
		    dr[2] = r[i].z - r[j].z;
            
		    dr[0] -= floorf((dr[0] / L) + 0.5f) * L;
		    dr[1] -= floorf((dr[1] / L) + 0.5f) * L;
		    dr[2] -= floorf((dr[2] / L) + 0.5f) * L;
                
                for (d2=0.0f, axis=0; axis<DIM; axis++) d2 += dr[axis] * dr[axis];
            
            if (d2 < Rcut * Rcut){
                    
                d2inv = 1.0f / d2;
                d6inv = d2inv * d2inv * d2inv;
                f = 24.0f * d2inv * d6inv * (2.0f * d6inv -1.0f);
                        
                        a[i].x += f * dr[0];
                        a[i].y += f * dr[1];
                        a[i].z += f * dr[2];
                        
                        a[i].w += 4.0f * d6inv * (d6inv - 1.0f) - eCut;
               
                } /* endif d2 */
            } /* endfor j */
	} /* endfor i */
}

/****************************************************************************************/

void Rescale (M_double4 *v, double T, int Natoms){
    
    int i;
    float vSum2=0.0f, fs;
    
    for (i=0;i<Natoms;i++) vSum2 += (v[i].x * v[i].x) + (v[i].y * v[i].y) + (v[i].z * v[i].z);
    
    fs = sqrt (3.0f * T * Natoms / vSum2); /* scaling factor to set the temperature */
    
    for (i=0;i<Natoms;i++){
        
        v[i].x *= fs;
        v[i].y *= fs;
        v[i].z *= fs;
    }
}

/****************************************************************************************/
/* The function initializes the r (fcc lattice), v, a arrays and sets c.o.m. speed to zero */

void Initialization (M_double4 *r, M_double4 *v, M_double4 *a, float T, int *seed, int Natoms, float rho){
    
    int i, axis, k, M=1, nX, nY, nZ;
    float vSum[DIM]={0.0f};
    float L = pow(Natoms / rho, 1.0f/3); // total box lenght
    float firstCell[4][3] = {
        {0.25f, 0.25f, 0.25f},
        {0.75f, 0.75f, 0.25f},
        {0.75f, 0.25f, 0.75f},
        {0.25f, 0.75f, 0.75f}};
    
    while (4 * M * M * M < Natoms) M++; // M^3 will be the nuber of boxes to contain all the Natoms
    
    float l = L / M; // is the single box lenght
    
    int n = 0;
    
    
    for (nX=0; nX<M;nX++)
        
        for (nY=0; nY<M;nY++)
            
            for (nZ=0; nZ<M;nZ++)
                
                for (k=0; k<4; k++)
                    
                    if (n<Natoms){
                        
                        r[n].x = (nX + firstCell[k][0]) * l;
                        r[n].y = (nY + firstCell[k][1]) * l;
                        r[n].z = (nZ + firstCell[k][2]) * l;
                        
                        n++;
                    }
    
    
    for (i=0;i<Natoms;i++){
        
        v[i].x = Gaussian();
        v[i].y = Gaussian();
        v[i].z = Gaussian();
        
        vSum[0] += v[i].x;
        vSum[1] += v[i].y;
        vSum[2] += v[i].z;
        
    }
    
    for (axis=0;axis<DIM;axis++) vSum[axis] /= Natoms;
    
    for (i=0;i<Natoms;i++){
        
        v[i].x -= vSum[0]; /* total momentum = NULL */
        v[i].y -= vSum[1];
        v[i].z -= vSum[2];
        
        a[i].x = 0.0f;
        a[i].y = 0.0f;
        a[i].z = 0.0f;
    }
    
    Rescale (v, T, Natoms);

}
/****************************************************************************************/

void Evolution (M_double4 *r, M_double4 *v, M_double4 *a, int Natoms, float rho){
    
    dim3 dimGrid (NUM_BLOCK, 1, 1);
    dim3 dimBlock (NUM_THREAD, 1, 1);
    
    Position <<<dimGrid, dimBlock>>>(r, v, a, Natoms);
    Acceleration <<<dimGrid, dimBlock>>>(r, a, Natoms, rho);
    HalfStep <<<dimGrid, dimBlock>>>(v, a, Natoms);
    
}
/****************************************************************************************/

int main () {
    
    int stepCount, stepLimit, seed, Natoms;
    float T, mtime;
    float rho;
    long seconds, useconds;
    M_double4 *h_r, *h_v, *h_a;
    M_double4 *d_r, *d_v, *d_a;
    struct timeval start, end;
    hipEvent_t gpu_start, gpu_stop;
    float gpu_runtime;
    FILE *fp;
    
    seed = time(0);
    srand48(seed);
  
    fp = fopen("cuda_time.dat","w+");
    
    stepLimit = 100;
    T = 0.5f;
    rho = 0.2f;
    
    while (rho<1.0f){
    
        for (Natoms=10;Natoms<=1500;Natoms+=100){
    
            size_t size = NUM_BLOCK * NUM_THREAD * sizeof ( M_double4 );
    
        
            /* allocating memory on the host */
            h_r = ( M_double4 * ) malloc ( size );
            h_v = ( M_double4 * ) malloc ( size );
            h_a = ( M_double4 * ) malloc ( size );
    
            /* allocating memory on the device */
            hipMalloc (&d_r, size);
            hipMalloc (&d_v, size);
            hipMalloc (&d_a, size);
    
            Initialization (h_r, h_v, h_a, T, &seed, Natoms, rho);
     
            /* copying data from host to device */
            hipMemcpy (d_r, h_r, size, hipMemcpyHostToDevice);
            hipMemcpy (d_v, h_v, size, hipMemcpyHostToDevice);
            hipMemcpy (d_a, h_a, size, hipMemcpyHostToDevice);
        
            gettimeofday (&start, NULL);
            
            hipEventCreate (&gpu_start);
            hipEventCreate (&gpu_stop);
            hipEventRecord (gpu_start, 0);
        
            /* main cycle */
            for (stepCount=0; stepCount<=stepLimit; stepCount++){
           
                Evolution (d_r, d_v, d_a, Natoms, rho);
            }
            
            hipEventRecord (gpu_stop, 0);
            hipEventSynchronize (gpu_stop);
            hipEventElapsedTime (&gpu_runtime, gpu_start, gpu_stop);
    
            gettimeofday (&end, NULL);
            seconds  = end.tv_sec  - start.tv_sec;
            useconds = end.tv_usec - start.tv_usec;
            mtime = (float)seconds + (float)useconds/1000000.0f;
        
            fprintf (fp, "%d\t%.5lf\n", Natoms, gpu_runtime /*1000 * mtime*/ / (stepLimit + 1));
    
            free (h_r); // freeing host memory
            free (h_v);
            free (h_a);
        
            hipFree (d_r); // freeing device memory
            hipFree (d_v);
            hipFree (d_a);

        }
    
        fprintf (fp,"\n\n");
        rho += 0.5f;
    }
	
        printf ("\nComputational times in file cuda_time.dat\n", mtime);

        fclose(fp);
    
        
}
/****************************************************************************************/
